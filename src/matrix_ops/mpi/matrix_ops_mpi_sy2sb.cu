#include "hip/hip_runtime.h"
#include <mpi.h>
#include <nccl.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cstddef>
#include <stdexcept>
#include <algorithm>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <sstream>

#include "fmt/format.h"
#include "log.h"
#include "matrix_ops.cuh"
#include "matrix_ops_mpi.cuh"
#include "sy2sb_panelqr.cuh"

namespace matrix_ops {
namespace mpi {

// 构造函数实现
MpiConfig::MpiConfig(int r, int s, int local_gpu, int total)
    : rank(r), size(s), local_gpu_id(local_gpu), total_gpus(total) {}

// MpiSy2sbContext 类方法实现
template <typename T>
MpiSy2sbContext<T>::MpiSy2sbContext(const MpiConfig& config, size_t matrix_n,
                                    T* A, size_t lda_val, T* W, size_t ldw_val,
                                    T* Y, size_t ldy_val, size_t nb_val,
                                    size_t b_val)
    : mpi_config(config),
      n(matrix_n),
      lda(lda_val),
      ldw(ldw_val),
      ldy(ldy_val),
      nb(nb_val),
      b(b_val),
      A_host(A),
      W_host(W),
      Y_host(Y) {
    // 默认使用现有 Blockwise 策略；支持通过环境变量预先切换为 BlockCyclic1D
    dist_type = DistributionType::Blockwise;
    block_size_bs = nb;  // block-cyclic 缺省使用 nb
    if (const char* dist_env = std::getenv("EVD_DIST")) {
        if (!std::strcmp(dist_env, "cyclic") ||
            !std::strcmp(dist_env, "blockcyclic") || !std::strcmp(dist_env, "bc")) {
            dist_type = DistributionType::BlockCyclic1D;
        }
    }

    // 计算分块信息（Blockwise）
    cols_per_process = n / mpi_config.size;
    start_col = mpi_config.rank * cols_per_process;
    local_matrix_size = cols_per_process * n;
    // 预计算当前策略下本地列数
    local_cols = computeLocalCols();
    if constexpr (std::is_same_v<T, float>) {
        nccl_type = ncclFloat32;
    } else if constexpr (std::is_same_v<T, double>) {
        nccl_type = ncclFloat64;
    }

    initGpuResources();
    initCommunication();
    allocateGpuMemory();
}

template <typename T>
MpiSy2sbContext<T>::~MpiSy2sbContext() {
    cleanup();
}

// 工具函数：计算给定列偏移对应的MPI进程
template <typename T>
size_t MpiSy2sbContext<T>::computeProcessForColumn(size_t col_offset) const {
    if (dist_type == DistributionType::Blockwise) {
        return col_offset / cols_per_process;
    }
    return ownerOfCol(col_offset);
}

// 工具函数：判断给定列是否属于当前进程
template <typename T>
bool MpiSy2sbContext<T>::isLocalColumn(size_t col_offset) const {
    return computeProcessForColumn(col_offset) ==
           static_cast<size_t>(mpi_config.rank);
}

// 工具函数：获取本地列索引
template <typename T>
size_t MpiSy2sbContext<T>::getLocalColumnIndex(size_t global_col) const {
    if (!isLocalColumn(global_col)) {
        throw std::out_of_range("Column is not local to this process");
    }
    // 与分布策略绑定
    if (dist_type == DistributionType::Blockwise) {
        return global_col - start_col;
    }
    return localColIndex(global_col);
}

template <typename T>
void MpiSy2sbContext<T>::initGpuResources() {
    // 设置当前进程使用的 GPU
    hipSetDevice(mpi_config.local_gpu_id);

    // 创建 CUDA 流
    hipStreamCreate(&stream);

    // 设置 cuBLAS 和 cuSOLVER 句柄的流
    hipblasSetStream(cublas_handle, stream);
    hipsolverSetStream(cusolver_handle, stream);
}

template <typename T>
void MpiSy2sbContext<T>::initCommunication() {
    // 在 MPI 环境中初始化层次化 NCCL 通信组
    // 获取所有进程的 GPU ID
    std::vector<int> all_gpu_ids(mpi_config.size);

    // 收集所有进程的本地 GPU ID
    MPI_Allgather(&mpi_config.local_gpu_id, 1, MPI_INT, all_gpu_ids.data(), 1,
                  MPI_INT, MPI_COMM_WORLD);

    // 创建层次化通信组：[0,1,2,3] -> [1,2,3] -> [2,3] -> [3]
    // 这样设计可以让早完成的进程开始下一轮计算，实现流水线并行

    // 1. 主通信组：所有进程 [0,1,2,3,...,size-1]
    ncclUniqueId main_nccl_id;
    if (mpi_config.rank == 0) {
        ncclGetUniqueId(&main_nccl_id);
    }
    MPI_Bcast(&main_nccl_id, sizeof(main_nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD);

    ncclResult_t nccl_result = ncclCommInitRank(&nccl_comm, mpi_config.size,
                                                main_nccl_id, mpi_config.rank);
    if (nccl_result != ncclSuccess) {
        throw std::runtime_error(
            fmt::format("Main NCCL initialization failed: {}",
                        ncclGetErrorString(nccl_result)));
    }

    // 2. 创建子通信组：每个进程参与从自己开始到最后的通信组
    // 进程i参与通信组 [i, i+1, ..., size-1]
    sub_comm_groups.resize(mpi_config.size);
    sub_mpi_comms.resize(mpi_config.size);  // 同时创建对应的MPI子通信器

    for (int start_rank = 0; start_rank < mpi_config.size; start_rank++) {
        // 所有进程都参与MPI_Comm_split，但只有部分进程会被分配到有效的通信器
        int color =
            (mpi_config.rank >= start_rank) ? start_rank : MPI_UNDEFINED;

        MPI_Comm_split(MPI_COMM_WORLD, color, mpi_config.rank,
                       &sub_mpi_comms[start_rank]);

        if (mpi_config.rank >= start_rank) {
            int sub_group_size = mpi_config.size - start_rank;
            int sub_rank = mpi_config.rank - start_rank;

            // 只有当子组大小大于1时才创建NCCL通信器
            // 单进程组不需要NCCL通信
            if (sub_group_size > 1) {
                // 生成该子组的NCCL ID
                ncclUniqueId sub_nccl_id;
                if (mpi_config.rank == start_rank) {
                    ncclGetUniqueId(&sub_nccl_id);
                }

                // 在子组内广播NCCL ID

                MPI_Bcast(&sub_nccl_id, sizeof(sub_nccl_id), MPI_BYTE, 0,
                          sub_mpi_comms[start_rank]);

                // 初始化子组NCCL通信器

                ncclResult_t sub_result =
                    ncclCommInitRank(&sub_comm_groups[start_rank],
                                     sub_group_size, sub_nccl_id, sub_rank);

                if (sub_result != ncclSuccess) {
                    throw std::runtime_error(fmt::format(
                        "Sub NCCL group {} initialization failed: {}",
                        start_rank, ncclGetErrorString(sub_result)));
                }

            } else {
                // 单进程组：不需要NCCL通信器
                sub_comm_groups[start_rank] = nullptr;
            }

        } else {
            sub_comm_groups[start_rank] = nullptr;
            // sub_mpi_comms[start_rank]
            // 已经在MPI_Comm_split中设置为MPI_COMM_NULL
        }
    }
}

template <typename T>
void MpiSy2sbContext<T>::allocateGpuMemory() {
    // 计算矩阵分布：
    // Blockwise 要求 n 能被 size 整除；BlockCyclic1D 无此硬性要求
    if (dist_type == DistributionType::Blockwise) {
        if (n % mpi_config.size != 0) {
            throw std::runtime_error(
                "Matrix size must be divisible by MPI size (blockwise)");
        }
    }

    // 设置当前 GPU 设备
    hipSetDevice(mpi_config.local_gpu_id);

    // 分配各个矩阵的 GPU 内存 (直接调用 resize)
    // A, W, Y, oriA: 存储本地矩阵块，按当前分布策略打包为连续列
    size_t cols_owned = (dist_type == DistributionType::Blockwise)
                            ? cols_per_process
                            : local_cols;
    size_t local_elems = cols_owned * n;
    gpu_A.resize(local_elems);
    gpu_W.resize(local_elems);
    gpu_Y.resize(local_elems);
    gpu_oriA.resize(local_elems);  // 原始矩阵 A 的备份

    // R: 存储 QR 分解的上三角矩阵 (n × nb)
    gpu_R.resize(n * nb);

    // Z: 工作矩阵，用于 Householder 向量 (n × nb)
    gpu_Z.resize(n * nb);

    // work: 临时工作空间 (2 × n × nb)
    gpu_work.resize(2 * n * nb);

    // 初始化为 0
    thrust::fill(gpu_W.begin(), gpu_W.end(), T(0));
    thrust::fill(gpu_Y.begin(), gpu_Y.end(), T(0));
    thrust::fill(gpu_Z.begin(), gpu_Z.end(), T(0));

    // 复制主机数据到 GPU
    copyHostToGpu();
}

template <typename T>
void MpiSy2sbContext<T>::copyHostToGpu() {
    try {
        if (dist_type == DistributionType::Blockwise) {
            // 按列复制 A 矩阵的本地连续部分
            thrust::copy(A_host + start_col * n,
                         A_host + start_col * n + local_matrix_size,
                         gpu_A.begin());
            // 同时复制到 oriA 作为原始矩阵的备份
            thrust::copy(A_host + start_col * n,
                         A_host + start_col * n + local_matrix_size,
                         gpu_oriA.begin());
        } else {
            // BlockCyclic1D: 逐列打包复制
            auto bs = block_size_bs;
            size_t local_idx = 0;
            for (size_t j = 0; j < n; ++j) {
                if (ownerOfCol(j) == static_cast<size_t>(mpi_config.rank)) {
                    // 列 j 的本地列号
                    size_t lc = localColIndex(j);
                    // 源列起始指针（列主序）
                    const T* src = A_host + j * n;
                    // 目标列起始指针（列主序，打包）
                    thrust::device_ptr<T> dst = gpu_A.data() + lc * n;
                    // 拷贝 n 元素（整列）
                    thrust::copy(src, src + n, dst);
                    // 备份 oriA
                    thrust::device_ptr<T> dst_ori = gpu_oriA.data() + lc * n;
                    thrust::copy(src, src + n, dst_ori);
                    local_idx += 1;
                }
            }
        }
    } catch (const std::exception& e) {
        throw std::runtime_error(
            fmt::format("Failed to copy host data to GPU: {}", e.what()));
    }
}

template <typename T>
void MpiSy2sbContext<T>::cleanup() {
    // 销毁所有子 NCCL 通信器
    for (size_t i = 0; i < sub_comm_groups.size(); i++) {
        if (sub_comm_groups[i] != nullptr) {
            ncclCommDestroy(sub_comm_groups[i]);
        }
    }
    sub_comm_groups.clear();

    // 销毁所有子 MPI 通信器
    for (size_t i = 0; i < sub_mpi_comms.size(); i++) {
        if (sub_mpi_comms[i] != MPI_COMM_NULL) {
            MPI_Comm_free(&sub_mpi_comms[i]);
        }
    }
    sub_mpi_comms.clear();

    // 销毁主 NCCL 通信器
    if (nccl_comm != nullptr) {
        ncclCommDestroy(nccl_comm);
    }

    // 销毁 CUDA 流
    hipStreamDestroy(stream);
}

// 计算 block-cyclic (1D) 下本地列数
template <typename T>
size_t MpiSy2sbContext<T>::computeLocalCols() const {
    if (dist_type == DistributionType::Blockwise) {
        return cols_per_process;
    }
    auto P = static_cast<size_t>(mpi_config.size);
    auto r = static_cast<size_t>(mpi_config.rank);
    auto bs = block_size_bs > 0 ? block_size_bs : nb;
    size_t full_blocks = n / bs;
    size_t tail = n % bs;

    size_t base_blocks = full_blocks / P;
    size_t extra_blocks = (r < (full_blocks % P)) ? 1 : 0;
    size_t cols = (base_blocks + extra_blocks) * bs;
    // 处理尾块
    if (tail > 0 && (full_blocks % P) == r) {
        cols += tail;
    }
    return cols;
}

namespace internal {

// 前向声明：调试用 CUDA 同步打印
static inline void debug_cuda_sync(const char* where);

// 定义：调试用 CUDA 同步打印（放在相同命名空间内，避免链接问题）
static inline void debug_cuda_sync(const char* where) {
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        util::MpiLogger::error("[CUDA] {}: {}", where,
                               hipGetErrorString(err));
    }
}

/**
 * @brief Functor to copy the lower triangular part of a matrix to the upper
 * triangular part.
 *
 * @tparam T Data type of the matrix elements.
 */
template <typename T>
struct make_symmetric_functor {
    thrust::device_ptr<T> A_;
    size_t n_;
    size_t lda_;

    make_symmetric_functor(thrust::device_ptr<T> A, size_t n, size_t lda)
        : A_(A), n_(n), lda_(lda) {}

    __device__ void operator()(const size_t& k) const {
        size_t j = k % n_;  // row
        size_t i = k / n_;  // col
        if (j < i) {
            A_[j + i * lda_] = A_[i + j * lda_];
        }
    }
};

// 新提取的函数，用于执行面板QR分解和相关的数据复制
template <typename T>
void performPanelQrComputeWy(int rank, const common::CublasHandle& handle,
                             const common::CusolverDnHandle& cusolver_handle,
                             size_t gpu_index, size_t panel_m, size_t panel_n,
                             thrust::device_ptr<T> panel_ptr, size_t lda,
                             thrust::device_ptr<T> R, size_t ldr,
                             thrust::device_ptr<T> panel_W_ptr, size_t ldw,
                             thrust::device_ptr<T> panel_Y_ptr, size_t ldy,
                             MPI_Comm& comm) {
    if (rank == gpu_index) {
        // execute panel QR decomposition
        matrix_ops::internal::sy2sb::panelQR(handle, cusolver_handle, panel_m,
                                             panel_n, panel_ptr, lda, R, ldr,
                                             panel_W_ptr, ldw);
        // copy panel data to panelY (using lda)
        matrix_ops::matrix_copy<thrust::device_ptr<T>, thrust::device_ptr<T>,
                                T>(panel_ptr, lda, panel_Y_ptr, ldy, panel_m,
                                   panel_n);

        // copy panelR data to panel (using lda)
        matrix_ops::matrix_copy<thrust::device_ptr<T>, thrust::device_ptr<T>,
                                T>(R, lda, panel_ptr, lda, panel_m, panel_n);
    }
    MPI_Barrier(comm);
}

template <typename T>
void performComputeAw(matrix_ops::mpi::MpiSy2sbContext<T>& ctx, MPI_Comm& comm,
                      int rank, size_t gpu_index, size_t panel_m,
                      size_t panel_n, size_t i, size_t lda, size_t ldw,
                      size_t ldz, size_t recrusive_offset,
                      size_t recrusive_offset_finished) {
    if (ctx.dist_type == DistributionType::Blockwise) {
        auto rest_gpu_num = ctx.mpi_config.size - gpu_index;

        // single card
        if (rest_gpu_num == 1) {
            auto oriA_panel = ctx.gpu_oriA.data() + recrusive_offset -
                              ctx.start_col * ctx.n + i * lda + i;
            auto panel_W_ptr = ctx.gpu_W.data() + recrusive_offset -
                               ctx.start_col * ctx.n + i + (i - ctx.b) * ldw;
            auto panel_Z_ptr = ctx.gpu_Z.data() + i + (i - ctx.b) * ldz;
            matrix_ops::gemm(ctx.cublas_handle, panel_m, ctx.b, panel_m, (T)1,
                             oriA_panel, lda, false, panel_W_ptr, ldw, false,
                             (T)0, panel_Z_ptr, ldz);
        } else {
            if (ctx.mpi_config.rank == gpu_index) {
                // copy W to workspace
                auto panel_W_ptr = ctx.gpu_W.data() + recrusive_offset -
                                   ctx.start_col * ctx.n + i + (i - ctx.b) * ldw;
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    panel_W_ptr, ldw, ctx.gpu_work.data(), panel_m, panel_m,
                    ctx.b);
            }

            // 使用子通信组进行广播
            auto& sub_comm = ctx.sub_comm_groups[gpu_index];
            if (sub_comm != nullptr) {
                // root进程在子通信组中的rank是0
                ncclBcast(ctx.gpu_work.data().get(), ctx.b * panel_m,
                          ctx.nccl_type, 0, sub_comm, ctx.stream);
                hipStreamSynchronize(ctx.stream);
            }

            auto oriA_panel =
                ctx.gpu_oriA.data() + i + recrusive_offset_finished;
            auto z_panel_rows = ctx.cols_per_process;

            if (gpu_index == ctx.mpi_config.rank) {
                oriA_panel = ctx.gpu_oriA.data() + recrusive_offset -
                             ctx.start_col * ctx.n + i + i * lda;
                z_panel_rows =
                    panel_m - ctx.cols_per_process * (rest_gpu_num - 1);
            }

            auto aw_panel = ctx.gpu_work.data() + ctx.n * ctx.nb;

            if (z_panel_rows > 0) {
                try {
                    matrix_ops::gemm(ctx.cublas_handle, z_panel_rows, ctx.b,
                                     panel_m, (T)1, oriA_panel, lda, true,
                                     ctx.gpu_work.data(), panel_m, false, (T)0,
                                     aw_panel, z_panel_rows);
                } catch (const std::exception& e) {
                    throw std::runtime_error(fmt::format(
                        "here aw gemm error exception: {}", e.what()));
                } catch (...) {
                    throw std::runtime_error(
                        "here aw gemm error: an unknown exception "
                        "occurred");
                }
            }

            std::vector<thrust::device_vector<T>> z_recv(rest_gpu_num - 1);

            // 使用子通信组进行Send/Recv
            if (sub_comm != nullptr) {
                ncclGroupStart();
                if (rank != gpu_index) {
                    // 在子通信组中，目标进程的rank是0
                    ncclSend(aw_panel.get(), ctx.cols_per_process * ctx.b,
                             ctx.nccl_type, 0, sub_comm, ctx.stream);
                } else {
                    for (auto gpu_offset = 1; gpu_offset < rest_gpu_num;
                         gpu_offset++) {
                        z_recv[gpu_offset - 1].resize(ctx.cols_per_process *
                                                      ctx.b);
                        // 在子通信组中，源进程的rank是 gpu_offset
                        ncclRecv(z_recv[gpu_offset - 1].data().get(),
                                 ctx.cols_per_process * ctx.b, ctx.nccl_type,
                                 gpu_offset, sub_comm, ctx.stream);
                    }
                }
                ncclGroupEnd();
            }

            hipStreamSynchronize(ctx.stream);

            if (rank == gpu_index) {
                auto panel_Z_ptr = ctx.gpu_Z.data() + i + (i - ctx.b) * ldz;
                if (z_panel_rows > 0) {
                    matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                            thrust::device_ptr<T>, T>(
                        aw_panel, z_panel_rows, panel_Z_ptr, ldz, z_panel_rows,
                        ctx.b);
                }
                for (auto index = 1; index < rest_gpu_num; index++) {
                    auto row_finished = (index - 1) * ctx.cols_per_process +
                                        panel_m - ctx.cols_per_process *
                                                       (rest_gpu_num - 1);
                    matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                            thrust::device_ptr<T>, T>(
                        z_recv[index - 1].data(), ctx.cols_per_process,
                        panel_Z_ptr + row_finished, ldz,
                        ctx.cols_per_process, ctx.b);
                }
            }

            MPI_Barrier(comm);
        }
        return;
    }

    // BlockCyclic1D: 基于循环块按 bs 大小分块处理并由块拥有者计算对应 Z 的行块
    auto bs = ctx.block_size_bs;
    size_t tail_start = recrusive_offset_finished + i;
    size_t panel_rows = panel_m;  // = n - tail_start

    // root 复制 W 到工作区并全员广播（使用主通信器，覆盖所有参与者）
    if (ctx.mpi_config.rank == gpu_index) {
        auto panel_W_ptr = ctx.ptrLocalRC(ctx.gpu_W, tail_start,
                                          recrusive_offset_finished + i -
                                              ctx.b);
        matrix_ops::matrix_copy<thrust::device_ptr<T>, thrust::device_ptr<T>,
                                T>(panel_W_ptr, ldw, ctx.gpu_work.data(),
                                   panel_rows, panel_rows, ctx.b);
    }

    ncclBcast(ctx.gpu_work.data().get(), ctx.b * panel_rows, ctx.nccl_type,
              gpu_index, ctx.nccl_comm, ctx.stream);
    hipStreamSynchronize(ctx.stream);

    // 每个循环块独立计算并装配到 owner 的 Z 中
    size_t num_blocks = (panel_rows + bs - 1) / bs;
    for (size_t t = 0; t < num_blocks; ++t) {
        size_t j0 = tail_start + t * bs;               // 该循环块的起始全局列
        size_t w = std::min(bs, ctx.n - j0);           // 块宽
        size_t owner = ctx.ownerOfCol(j0);             // 块拥有者（按循环块）
        bool i_am_owner = (ctx.mpi_config.rank == static_cast<int>(owner));

        // 拿到本地 A 子块起点（panel_rows x w），按列主序打包
        thrust::device_ptr<T> A_block;
        if (i_am_owner) {
            // 注意行起点应为 tail_start，而不是 j0
            A_block = ctx.ptrLocalRC(ctx.gpu_oriA, tail_start, j0);
        }

        // 计算 aw_block = A_block^T * W  => 尺寸 (w x b)，列主序，ld = w
        thrust::device_ptr<T> aw_block = ctx.gpu_work.data() + ctx.n * ctx.nb;
        if (i_am_owner) {
            if (w > 0) {
                matrix_ops::gemm(ctx.cublas_handle, w, ctx.b, panel_rows,
                                 (T)1, A_block, lda, true, ctx.gpu_work.data(),
                                 panel_rows, false, (T)0, aw_block, w);
            }
        }

        // 非 owner 不计算，直接同步
        hipStreamSynchronize(ctx.stream);

        // 把结果送到 panel owner（gpu_index）并由其写入 Z 相应行偏移
        size_t row_offset = t * bs;  // 该块在 Z 中的起始行（按全局列顺序）
        if (ctx.mpi_config.rank != gpu_index) {
            if (i_am_owner && w > 0) {
                ncclSend(aw_block.get(), w * ctx.b, ctx.nccl_type, gpu_index,
                         ctx.nccl_comm, ctx.stream);
            }
        }

        if (ctx.mpi_config.rank == gpu_index) {
            // 在 owner 上：接收或直接使用本地计算的块，然后拷贝到 Z
            thrust::device_vector<T> recv_buf;  // 按需分配
            thrust::device_ptr<T> src_block = aw_block;
            if (!i_am_owner) {
                if (w > 0) {
                    recv_buf.resize(w * ctx.b);
                    ncclRecv(recv_buf.data().get(), w * ctx.b, ctx.nccl_type,
                             static_cast<int>(owner), ctx.nccl_comm,
                             ctx.stream);
                    src_block = recv_buf.data();
                }
            }

            auto panel_Z_ptr = ctx.ptrLocalRC(
                ctx.gpu_Z, tail_start, recrusive_offset_finished + i - ctx.b);
            if (w > 0) {
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    src_block, w, panel_Z_ptr + row_offset, ldz, w, ctx.b);
            }
        }

        hipStreamSynchronize(ctx.stream);
        MPI_Barrier(comm);
    }
}

template <typename T>
void performInterRecursiveSyr2k(size_t recrusive_depth,
                                matrix_ops::mpi::MpiSy2sbContext<T>& ctx,
                                size_t gpu_index, thrust::device_ptr<T> A,
                                size_t lda, thrust::device_ptr<T> Y, size_t ldy,
                                thrust::device_ptr<T> Z, size_t ldz) {
    auto offset = (recrusive_depth + 1) * (ctx.nb + ctx.nb * ctx.n);
    auto tail_gpu_start_index =
        ctx.computeProcessForColumn((recrusive_depth + 1) * ctx.nb);
    auto rest_gpu_num = ctx.mpi_config.size - gpu_index;
    auto sub_matrix_n = ctx.n - (recrusive_depth + 1) * ctx.nb;
    MPI_Comm comm = (ctx.dist_type == DistributionType::BlockCyclic1D)
                        ? MPI_COMM_WORLD
                        : ctx.sub_mpi_comms[gpu_index];
    auto tail_start = (recrusive_depth + 1) * ctx.nb;

    // BlockCyclic1D: 采用循环块分配尾部更新，每个块的列由其拥有者本地更新
    if (ctx.dist_type == DistributionType::BlockCyclic1D) {
        // 1) 准备并广播 Y_tail 与 Z_tail（尺寸 sub_n x nb，按 ld=sub_n 打包）
        auto sub_n = sub_matrix_n;
        auto bs = ctx.block_size_bs;

        thrust::device_ptr<T> y_bcast = ctx.gpu_work.data();
        thrust::device_ptr<T> z_bcast = ctx.gpu_work.data() + ctx.n * ctx.nb;

        if (ctx.mpi_config.rank == gpu_index) {
            // 源在面板 owner 上：把 [row=tail_start: , col=panel_start] 打成连续
            auto y_src = ctx.ptrLocalRC(ctx.gpu_Y, tail_start, tail_start - ctx.nb);
            auto z_src = ctx.ptrLocalRC(ctx.gpu_Z, tail_start, tail_start - ctx.nb);
            matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                    thrust::device_ptr<T>, T>(
                y_src, ldy, y_bcast, sub_n, sub_n, ctx.nb);
            matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                    thrust::device_ptr<T>, T>(
                z_src, ldz, z_bcast, sub_n, sub_n, ctx.nb);
        }

        // 广播到所有进程
        ncclBcast(y_bcast.get(), sub_n * ctx.nb, ctx.nccl_type, gpu_index,
                  ctx.nccl_comm, ctx.stream);
        ncclBcast(z_bcast.get(), sub_n * ctx.nb, ctx.nccl_type, gpu_index,
                  ctx.nccl_comm, ctx.stream);
        hipStreamSynchronize(ctx.stream);

        // 2) 遍历尾部按 bs 分块的列块，由块拥有者计算并就地更新 C(:,J)
        size_t num_blocks = (sub_n + bs - 1) / bs;
        for (size_t t = 0; t < num_blocks; ++t) {
            size_t j0 = tail_start + t * bs;    // 全局列起点
            size_t w = std::min(bs, ctx.n - j0);  // 块宽
            size_t owner = ctx.ownerOfCol(j0);
            bool i_am_owner = (ctx.mpi_config.rank == static_cast<int>(owner));

            if (w == 0) continue;

            if (i_am_owner) {
                // 本地 C(:,J) 指针（在 oriA 中更新）和 A 镜像
                auto c_block = ctx.ptrLocalRC(ctx.gpu_oriA, tail_start, j0);
                auto a_block = ctx.ptrLocalRC(ctx.gpu_A, tail_start, j0);

                // 选取 Y/J 和 Z/J 的行块（w x nb），并进行两个 GEMM 叠加
                size_t row_off = j0 - tail_start;  // 在打包 y/z 中的行偏移
                auto y_rows = y_bcast + row_off;   // (w x nb) with ld=sub_n
                auto z_rows = z_bcast + row_off;   // (w x nb) with ld=sub_n

                // C(:,J) -= Y * Z[J,:]^T
                matrix_ops::gemm(ctx.cublas_handle, sub_n, w, ctx.nb, T(-1),
                                 y_bcast, sub_n, false, z_rows, sub_n, true,
                                 T(1), c_block, lda);
                // C(:,J) -= Z * Y[J,:]^T
                matrix_ops::gemm(ctx.cublas_handle, sub_n, w, ctx.nb, T(-1),
                                 z_bcast, sub_n, false, y_rows, sub_n, true,
                                 T(1), c_block, lda);

                // 可选：保持 A 镜像一致（复制更新后的列块到 gpu_A）
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    c_block, lda, a_block, lda, sub_n, w);
            }

            hipStreamSynchronize(ctx.stream);
            MPI_Barrier(comm);
        }

        return;
    }
    thrust::device_ptr<T> tail_matrix_ptr;
    if (ctx.mpi_config.rank == tail_gpu_start_index) {
        tail_matrix_ptr = ctx.gpu_oriA.data() + offset - ctx.start_col * ctx.n;
    }
    thrust::device_vector<T> z_send;
    if (ctx.mpi_config.rank == gpu_index) {
        z_send.resize(ctx.n * ctx.nb);
    }
    if (gpu_index == tail_gpu_start_index) {
        if (rest_gpu_num == 1) {
            if (ctx.mpi_config.rank == gpu_index) {
                matrix_ops::syr2k(ctx.cublas_handle, sub_matrix_n, ctx.nb,
                                  (T)(-1), Y + ctx.nb, ldy, Z + ctx.nb, ldz,
                                  (T)1, tail_matrix_ptr, lda);

                thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                                 thrust::make_counting_iterator<size_t>(
                                     sub_matrix_n * sub_matrix_n),
                                 make_symmetric_functor<T>(tail_matrix_ptr,
                                                           sub_matrix_n, lda));

                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    tail_matrix_ptr, ctx.n, A + ctx.nb + ctx.nb * lda, ctx.n,
                    sub_matrix_n, sub_matrix_n);
            }
        } else {
            if (ctx.mpi_config.rank == gpu_index) {
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Y + ctx.nb, lda, ctx.gpu_work.data(), sub_matrix_n,
                    sub_matrix_n, ctx.nb);
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Z + ctx.nb, lda, z_send.data(), sub_matrix_n, sub_matrix_n,
                    ctx.nb);
            }
            MPI_Barrier(comm);

            auto z_bcast = ctx.gpu_Z.data();
            if (ctx.mpi_config.rank == gpu_index) {
                z_bcast = z_send.data();
            }
            auto& sub_comm = ctx.sub_comm_groups[gpu_index];
            if (sub_comm != nullptr) {
                ncclBcast(z_bcast.get(), sub_matrix_n * ctx.nb, ctx.nccl_type,
                          0, sub_comm, ctx.stream);
                ncclBcast(ctx.gpu_work.data().get(), sub_matrix_n * ctx.nb,
                          ctx.nccl_type, 0, sub_comm, ctx.stream);
            }
            hipStreamSynchronize(ctx.stream);

            auto syr2k_panel_col = ctx.cols_per_process;
            auto& syr2k_panel_handle = ctx.cublas_handle;
            auto syr2k_panel_oriA_ptr =
                ctx.gpu_oriA.data() + (ctx.n - sub_matrix_n);
            auto dst_A_ptr = ctx.gpu_A.data() + (ctx.n - sub_matrix_n);
            auto zy_panel_offset =
                sub_matrix_n - (ctx.mpi_config.size - ctx.mpi_config.rank) *
                                   ctx.cols_per_process;
            if (ctx.mpi_config.rank == gpu_index) {
                syr2k_panel_col =
                    sub_matrix_n - (rest_gpu_num - 1) * ctx.cols_per_process;
                syr2k_panel_oriA_ptr = tail_matrix_ptr;
                dst_A_ptr = ctx.gpu_A.data() + offset - ctx.start_col * ctx.n;
                zy_panel_offset = 0;
            }

            matrix_ops::gemm(syr2k_panel_handle, sub_matrix_n, syr2k_panel_col,
                             ctx.nb, T(-1), z_bcast, sub_matrix_n, false,
                             ctx.gpu_work.data() + zy_panel_offset,
                             sub_matrix_n, true, T(1), syr2k_panel_oriA_ptr,
                             lda);
            matrix_ops::gemm(syr2k_panel_handle, sub_matrix_n, syr2k_panel_col,
                             ctx.nb, T(-1), ctx.gpu_work.data(), sub_matrix_n,
                             false, z_bcast + zy_panel_offset, sub_matrix_n,
                             true, T(1), syr2k_panel_oriA_ptr, lda);

            matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                    thrust::device_ptr<T>, T>(
                syr2k_panel_oriA_ptr, lda, dst_A_ptr, lda, sub_matrix_n,
                syr2k_panel_col);
        }
    } else {
        if (rest_gpu_num == 1) {
            if (ctx.mpi_config.rank == gpu_index) {
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Y + ctx.nb, lda, ctx.gpu_work.data(), sub_matrix_n,
                    sub_matrix_n, ctx.nb);
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Z + ctx.nb, lda, z_send.data(), sub_matrix_n, sub_matrix_n,
                    ctx.nb);
            }

            ncclGroupStart();
            if (ctx.mpi_config.rank == gpu_index) {
                ncclSend(ctx.gpu_work.data().get(), sub_matrix_n * ctx.nb,
                         ctx.nccl_type, tail_gpu_start_index, ctx.nccl_comm,
                         ctx.stream);
            } else if (ctx.mpi_config.rank == tail_gpu_start_index) {
                ncclRecv(ctx.gpu_work.data().get(), sub_matrix_n * ctx.nb,
                         ctx.nccl_type, gpu_index, ctx.nccl_comm, ctx.stream);
            }
            ncclGroupEnd();

            hipStreamSynchronize(ctx.stream);

            ncclGroupStart();
            if (ctx.mpi_config.rank == gpu_index) {
                ncclSend(z_send.data().get(), sub_matrix_n * ctx.nb,
                         ctx.nccl_type, tail_gpu_start_index, ctx.nccl_comm,
                         ctx.stream);
            } else if (ctx.mpi_config.rank == tail_gpu_start_index) {
                ncclRecv(ctx.gpu_Z.data().get(), sub_matrix_n * ctx.nb,
                         ctx.nccl_type, gpu_index, ctx.nccl_comm, ctx.stream);
            }
            ncclGroupEnd();

            hipStreamSynchronize(ctx.stream);

            if (ctx.mpi_config.rank == tail_gpu_start_index) {
                matrix_ops::syr2k(ctx.cublas_handle, sub_matrix_n, ctx.nb,
                                  (T)(-1), ctx.gpu_work.data(), sub_matrix_n,
                                  ctx.gpu_Z.data(), sub_matrix_n, (T)1,
                                  tail_matrix_ptr, lda);
                thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                                 thrust::make_counting_iterator<size_t>(
                                     sub_matrix_n * sub_matrix_n),
                                 make_symmetric_functor<T>(tail_matrix_ptr,
                                                           sub_matrix_n, lda));
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    tail_matrix_ptr, lda,
                    ctx.gpu_A.data() + offset - ctx.cols_per_process * ctx.n,
                    lda, sub_matrix_n, sub_matrix_n);
            }
        } else {
            if (ctx.mpi_config.rank == gpu_index) {
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Y + ctx.nb, lda, ctx.gpu_work.data(), sub_matrix_n,
                    sub_matrix_n, ctx.nb);

                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    Z + ctx.nb, lda, z_send.data(), sub_matrix_n, sub_matrix_n,
                    ctx.nb);
            }
            auto z_bcast = ctx.gpu_Z.data();
            if (ctx.mpi_config.rank == gpu_index) {
                z_bcast = z_send.data();
            }
            auto& sub_comm = ctx.sub_comm_groups[gpu_index];
            if (sub_comm != nullptr) {
                ncclBcast(z_bcast.get(), sub_matrix_n * ctx.nb, ctx.nccl_type,
                          0, sub_comm, ctx.stream);
                ncclBcast(ctx.gpu_work.data().get(), sub_matrix_n * ctx.nb,
                          ctx.nccl_type, 0, sub_comm, ctx.stream);
            }
            hipStreamSynchronize(ctx.stream);
            if (ctx.mpi_config.rank != gpu_index) {
                auto syr2k_panel_col = ctx.cols_per_process;
                auto& syr2k_panel_handle = ctx.cublas_handle;
                auto syr2k_panel_oriA_ptr =
                    ctx.gpu_oriA.data() + (ctx.n - sub_matrix_n);
                auto dst_A_ptr = ctx.gpu_A.data() + (ctx.n - sub_matrix_n);

                auto zy_panel_offset =
                    sub_matrix_n - (ctx.mpi_config.size - ctx.mpi_config.rank) *
                                       ctx.cols_per_process;

                matrix_ops::gemm(
                    syr2k_panel_handle, sub_matrix_n, syr2k_panel_col, ctx.nb,
                    T(-1), z_bcast, sub_matrix_n, false,
                    ctx.gpu_work.data() + zy_panel_offset, sub_matrix_n, true,
                    T(1), syr2k_panel_oriA_ptr, lda);
                matrix_ops::gemm(syr2k_panel_handle, sub_matrix_n,
                                 syr2k_panel_col, ctx.nb, T(-1),
                                 ctx.gpu_work.data(), sub_matrix_n, false,
                                 z_bcast + zy_panel_offset, sub_matrix_n, true,
                                 T(1), syr2k_panel_oriA_ptr, lda);
                matrix_ops::matrix_copy<thrust::device_ptr<T>,
                                        thrust::device_ptr<T>, T>(
                    syr2k_panel_oriA_ptr, lda, dst_A_ptr, lda, sub_matrix_n,
                    syr2k_panel_col);
            }
        }
    }
    MPI_Barrier(comm);
}

template <typename T>
void sy2sb_recursive_mpi(size_t recursive_depth,
                         matrix_ops::mpi::MpiSy2sbContext<T>& ctx) {
    // compute recrusive offset and panel related resources
    auto recrusive_offset_finished = ctx.nb * recursive_depth;
    auto recrusive_offset = (ctx.nb + ctx.nb * ctx.n) * recursive_depth;
    auto gpu_index = ctx.computeProcessForColumn(recrusive_offset_finished);

    if (ctx.dist_type != DistributionType::BlockCyclic1D &&
        ctx.mpi_config.rank < gpu_index) {
        return;
    }

    MPI_Comm mpi_comm = (ctx.dist_type == DistributionType::BlockCyclic1D)
                            ? MPI_COMM_WORLD
                            : ctx.sub_mpi_comms[gpu_index];
    auto& handle = ctx.cublas_handle;
    auto& cusolver_handle = ctx.cusolver_handle;

    thrust::device_ptr<T> A, W, Y, Z, R, work_ptr;

    // 调试：进入递归同步一次，捕获前序非法访问
    internal::debug_cuda_sync("enter sy2sb_recursive_mpi");

    if (ctx.mpi_config.rank == gpu_index) {
        if (ctx.dist_type == DistributionType::Blockwise) {
            A = ctx.gpu_A.data() + recrusive_offset - ctx.start_col * ctx.n;
            W = ctx.gpu_W.data() + recrusive_offset - ctx.start_col * ctx.n;
            Y = ctx.gpu_Y.data() + recrusive_offset - ctx.start_col * ctx.n;
        } else {
            // block-cyclic: A/W/Y 直接使用映射在循环中计算的 panel 指针
            A = nullptr;
            W = nullptr;
            Y = nullptr;
        }
        R = ctx.gpu_R.data() + recrusive_offset_finished;
        Z = ctx.gpu_Z.data();
        work_ptr = ctx.gpu_work.data();
    }

    auto lda = ctx.n;
    auto ldw = ctx.n;
    auto ldr = ctx.n;
    auto ldy = ctx.n;
    auto ldz = ctx.n;
    auto ldwork = ctx.nb;

    // for-loop update with b panel
    for (auto i = ctx.b; i <= ctx.nb && i < ctx.n - recrusive_offset_finished;
         i += ctx.b) {
        // 迭代开始时同步一次，便于定位上一轮留下的非法访问
        if (ctx.dist_type == DistributionType::BlockCyclic1D) {
            std::ostringstream oss;
            oss << "enter iter depth=" << recursive_depth << " i=" << i
                << " owner=" << gpu_index << " rank="
                << ctx.mpi_config.rank;
            internal::debug_cuda_sync(oss.str().c_str());
        }
        thrust::device_ptr<T> panel_ptr, panel_W_ptr, panel_Y_ptr, panel_Z_ptr;
        auto panel_m = ctx.n - recrusive_offset_finished - i;
        auto panel_n = ctx.b;
        if (ctx.dist_type == DistributionType::Blockwise) {
            panel_ptr = A + i + (i - ctx.b) * lda;
            panel_W_ptr = W + i + (i - ctx.b) * ldw;
            panel_Y_ptr = Y + i + (i - ctx.b) * ldy;
            panel_Z_ptr = Z + i + (i - ctx.b) * ldz;
        } else {
            // block-cyclic: 仅在面板拥有者上计算面板指针
            if (ctx.mpi_config.rank == gpu_index) {
                panel_ptr = ctx.ptrLocalRC(
                    ctx.gpu_A, recrusive_offset_finished + i,
                    recrusive_offset_finished + i - ctx.b);
                panel_W_ptr = ctx.ptrLocalRC(
                    ctx.gpu_W, recrusive_offset_finished + i,
                    recrusive_offset_finished + i - ctx.b);
                panel_Y_ptr = ctx.ptrLocalRC(
                    ctx.gpu_Y, recrusive_offset_finished + i,
                    recrusive_offset_finished + i - ctx.b);
                panel_Z_ptr = ctx.ptrLocalRC(
                    ctx.gpu_Z, recrusive_offset_finished + i,
                    recrusive_offset_finished + i - ctx.b);

                // 校验面板所属与本地列索引
                size_t panel_col = recrusive_offset_finished + i - ctx.b;
                size_t owner = ctx.ownerOfCol(panel_col);
                if (owner != gpu_index) {
                    util::MpiLogger::error(
                        "[cyclic] panel owner mismatch: depth={} i={} expect_owner={} calc_owner={} rank={}",
                        recursive_depth, i, gpu_index, owner,
                        ctx.mpi_config.rank);
                }
                size_t lc = ctx.localColIndex(panel_col);
                if (lc >= ctx.local_cols) {
                    util::MpiLogger::error(
                        "[cyclic] localColIndex OOB: depth={} i={} lc={} local_cols={} rank={} col={}",
                        recursive_depth, i, lc, ctx.local_cols,
                        ctx.mpi_config.rank, panel_col);
                }
            }
        }

        internal::debug_cuda_sync("before panelQR");

        // process for this panel do the work
        try {
            performPanelQrComputeWy<T>(ctx.mpi_config.rank, handle,
                                       ctx.cusolver_handle, gpu_index, panel_m,
                                       panel_n, panel_ptr, lda, R, ldr,
                                       panel_W_ptr, ldw, panel_Y_ptr, ldy,
                                       mpi_comm);
        } catch (const std::exception& e) {
            util::MpiLogger::error(
                "panelQR failed at depth={} i={} owner={} rank={} m={} n={} err={}",
                recursive_depth, i, gpu_index, ctx.mpi_config.rank, panel_m,
                panel_n, e.what());
            debug_cuda_sync("panelQR");
            throw;
        }

        internal::debug_cuda_sync("after panelQR");

        // compute AW distribution
        try {
            performComputeAw<T>(ctx, mpi_comm, ctx.mpi_config.rank, gpu_index,
                                panel_m, panel_n, i, lda, ldw, ldz,
                                recrusive_offset, recrusive_offset_finished);
        } catch (const std::exception& e) {
            util::MpiLogger::error(
                "performComputeAw failed at depth={} i={} owner={} rank={} m={} n={} err={}",
                recursive_depth, i, gpu_index, ctx.mpi_config.rank, panel_m,
                panel_n, e.what());
            debug_cuda_sync("performComputeAw");
            throw;
        }

        internal::debug_cuda_sync("after performComputeAw");

        // compute all b panel update
        if (ctx.mpi_config.rank == gpu_index) {
            if (i == ctx.b) {
                try {
                    // panel_tmp = panel_z^T * panel_z
                    matrix_ops::gemm(handle, ctx.b, ctx.b, panel_m, (T)1,
                                     panel_W_ptr, ldw, true, panel_Z_ptr, ldz,
                                     false, (T)0, work_ptr, ldwork);
                    // panel_z = panel_z - panel_y * panel_z^T * panel_z
                    matrix_ops::gemm(handle, panel_m, ctx.b, ctx.b, (T)(-0.5),
                                     panel_Y_ptr, ldy, false, work_ptr, ldwork,
                                     false, (T)1, panel_Z_ptr, ldz);
                } catch (...) {
                    throw std::runtime_error(
                        "Error during initial panel update in sy2sb");
                }
            } else {
                try {
                    // panel_tmp = (Z + i)^T * panel_w
                    auto Z_ip = (ctx.dist_type == DistributionType::Blockwise)
                                    ? (Z + i)
                                    : ctx.ptrLocalRC(ctx.gpu_Z,
                                                     recrusive_offset_finished +
                                                         i,
                                                     recrusive_offset_finished +
                                                         i);
                    auto Y_ip = (ctx.dist_type == DistributionType::Blockwise)
                                    ? (Y + i)
                                    : ctx.ptrLocalRC(ctx.gpu_Y,
                                                     recrusive_offset_finished +
                                                         i,
                                                     recrusive_offset_finished +
                                                         i);
                    auto A_dst = (ctx.dist_type == DistributionType::Blockwise)
                                     ? (A + i + i * lda)
                                     : ctx.ptrLocalRC(
                                           ctx.gpu_A,
                                           recrusive_offset_finished + i,
                                           recrusive_offset_finished + i);

                    matrix_ops::gemm(handle, i - ctx.b, ctx.b, panel_m, (T)1,
                                     Z_ip, ldz, true, panel_W_ptr, ldw, false,
                                     (T)0, work_ptr, ldwork);
                    // panel_z = panel_z - Y+i * panel_z^T * panel_w
                    matrix_ops::gemm(handle, panel_m, ctx.b, i - ctx.b,
                                     (T)(-1), Y_ip, ldy, false, work_ptr,
                                     ldwork, false, (T)1, panel_Z_ptr, ldz);
                    // panel_tmp = Y+i^T * panel_w
                    matrix_ops::gemm(handle, i - ctx.b, ctx.b, panel_m,
                                     (T)(1), Y_ip, ldy, true, panel_W_ptr, ldw,
                                     false, (T)0, work_ptr, ldwork);
                    // panel_z = panel_z - (Z + i) * Y+i^T * panel_w
                    matrix_ops::gemm(handle, panel_m, ctx.b, i - ctx.b,
                                     (T)(-1), Z_ip, ldz, false, work_ptr,
                                     ldwork, false, (T)1, panel_Z_ptr, ldz);
                    // panel_tmp = panel_w^T * panel_z
                    matrix_ops::gemm(handle, ctx.b, ctx.b, panel_m, (T)1,
                                     panel_W_ptr, ldw, true, panel_Z_ptr, ldz,
                                     false, (T)0, work_ptr, ldwork);
                    // panel_z = panel_z - 0.5 * panel_y * panel_w^T * panel_z
                    matrix_ops::gemm(handle, panel_m, ctx.b, ctx.b, (T)(-0.5),
                                     panel_Y_ptr, ldy, false, work_ptr, ldwork,
                                     false, (T)1, panel_Z_ptr, ldz);
                } catch (...) {
                    throw std::runtime_error("Error in gemm");
                }
            }
            if (i < ctx.nb) {
                auto Z_ip = (ctx.dist_type == DistributionType::Blockwise)
                                ? (Z + i)
                                : ctx.ptrLocalRC(ctx.gpu_Z,
                                                 recrusive_offset_finished + i,
                                                 recrusive_offset_finished +
                                                     i);
                auto Y_ip = (ctx.dist_type == DistributionType::Blockwise)
                                ? (Y + i)
                                : ctx.ptrLocalRC(ctx.gpu_Y,
                                                 recrusive_offset_finished + i,
                                                 recrusive_offset_finished +
                                                     i);
                auto A_dst = (ctx.dist_type == DistributionType::Blockwise)
                                 ? (A + i + i * lda)
                                 : ctx.ptrLocalRC(ctx.gpu_A,
                                                  recrusive_offset_finished + i,
                                                  recrusive_offset_finished +
                                                      i);

                matrix_ops::gemm(handle, panel_m, ctx.b, i, (T)(-1), Y_ip, ldy,
                                 false, Z_ip, ldz, true, (T)1, A_dst, lda);

                matrix_ops::gemm(handle, panel_m, ctx.b, i, (T)(-1), Z_ip, ldz,
                                 false, Y_ip, ldy, true, (T)1, A_dst, lda);
            }
        }
        MPI_Barrier(mpi_comm);
    }

    // recursive quit
    if (ctx.n <= ctx.nb + recrusive_offset_finished) {
        return;
    }

    try {
        performInterRecursiveSyr2k(recursive_depth, ctx, gpu_index, A, lda, Y,
                                   ldy, Z, ldz);
    } catch (const std::exception& e) {
        util::MpiLogger::error(
            "performInterRecursiveSyr2k failed at depth={} owner={} rank={} err={}",
            recursive_depth, gpu_index, ctx.mpi_config.rank, e.what());
        debug_cuda_sync("performInterRecursiveSyr2k");
        throw;
    }

    // recursive call
    sy2sb_recursive_mpi(recursive_depth + 1, ctx);
}
}  // namespace internal

/**
 * @brief MPI 版本的 sy2sb 主函数
 */
template <typename T>
void sy2sb(const MpiConfig& mpi_config, size_t n, T* A, size_t lda, T* W,
           size_t ldw, T* Y, size_t ldy, size_t nb, size_t b) {
    // 检查分块兼容性（与分布式版本保持一致）
    if (n % b % mpi_config.size != 0) {
        throw std::runtime_error(
            "Matrix is not well divisible into MPI process panels");
    }

    // 创建 MPI sy2sb 上下文
    MpiSy2sbContext<T> ctx(mpi_config, n, A, lda, W, ldw, Y, ldy, nb, b);

    // 可通过环境变量切换分布策略：EVD_DIST=cyclic 使用 1D block-cyclic，默认 blockwise
    if (const char* dist_env = std::getenv("EVD_DIST")) {
        if (!std::strcmp(dist_env, "cyclic") || !std::strcmp(dist_env, "blockcyclic") ||
            !std::strcmp(dist_env, "bc")) {
            ctx.dist_type = DistributionType::BlockCyclic1D;
            // block_size_bs 默认等于 nb
        }
    }

    // 调试输出：确认当前分布模式和块大小（只在 rank 0 打印）
    if (mpi_config.rank == 0) {
        const char* mode = (ctx.dist_type == DistributionType::BlockCyclic1D)
                               ? "cyclic"
                               : "blockwise";
        util::MpiLogger::println("[sy2sb] distribution={}, bs={}, nb={}, b={}",
                                 mode, ctx.block_size_bs, nb, b);
    }

    util::MpiLogger::tic("sy2sb mpi");
    // 调用递归实现
    internal::sy2sb_recursive_mpi<T>(0, ctx);
    util::MpiLogger::toc("sy2sb mpi");

    // 最后全局同步确保所有进程完成
    MPI_Barrier(MPI_COMM_WORLD);
}

}  // namespace mpi

// 显式模板实例化
template class matrix_ops::mpi::MpiSy2sbContext<float>;
template class matrix_ops::mpi::MpiSy2sbContext<double>;

template void matrix_ops::mpi::sy2sb<float>(
    const matrix_ops::mpi::MpiConfig& mpi_config, size_t n, float* A,
    size_t lda, float* W, size_t ldw, float* Y, size_t ldy, size_t nb,
    size_t b);

template void matrix_ops::mpi::sy2sb<double>(
    const matrix_ops::mpi::MpiConfig& mpi_config, size_t n, double* A,
    size_t lda, double* W, size_t ldw, double* Y, size_t ldy, size_t nb,
    size_t b);

}  // namespace matrix_ops
