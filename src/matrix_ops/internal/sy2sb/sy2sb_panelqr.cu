#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include <stdexcept>

#include "gpu_handle_wrappers.h"
#include "matrix_ops.cuh"
#include "sy2sb_panelqr.cuh"

namespace matrix_ops {
namespace internal {
namespace sy2sb {

template <typename T>
void getIminusQL4panelQR(const common::CusolverDnHandle& handle, size_t m,
                         size_t n, thrust::device_ptr<T> A_inout, size_t lda) {
    auto lwork = (int)0;
    if constexpr (std::is_same_v<T, double>) {
        hipsolverDnDgetrf_bufferSize(
            handle, m, n, thrust::raw_pointer_cast(A_inout), lda, &lwork);
    } else if constexpr (std::is_same_v<T, float>) {
        hipsolverDnSgetrf_bufferSize(
            handle, m, n, thrust::raw_pointer_cast(A_inout), lda, &lwork);
    } else {
        throw std::runtime_error("Unsupported type.");
    }
    auto work = thrust::device_vector<T>(lwork);
    auto info = thrust::device_vector<int>(1);
    // excute LU factorization inplace
    // matrix_ops::print(A_inout, m, n, lda, "A_inout before getrf");
    if constexpr (std::is_same_v<T, double>) {
        hipsolverDnDgetrf(handle, m, n, thrust::raw_pointer_cast(A_inout), lda,
                         thrust::raw_pointer_cast(work.data()), NULL,
                         thrust::raw_pointer_cast(info.data()));
    } else if constexpr (std::is_same_v<T, float>) {
        hipsolverDnSgetrf(handle, m, n, thrust::raw_pointer_cast(A_inout), lda,
                         thrust::raw_pointer_cast(work.data()), NULL,
                         thrust::raw_pointer_cast(info.data()));
    } else {
        throw std::runtime_error("Unsupported type.");
    }
    auto info_host = thrust::host_vector<int>(info);
    if (info_host[0] != 0) {
        throw std::runtime_error("Failed to factorize the matrix.");
    }
    // matrix_ops::print(A_inout, m, n, lda, "A_inout before get L");

    // extract the lower triangular part of the matrix
    try {
        size_t num_elements = m * n;
        thrust::for_each(thrust::device, thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(num_elements),
                         extract_L_functor_2d<T>(A_inout, m, lda));
    } catch (std::exception& e) {
        throw std::runtime_error(fmt::format(
            "Error in panelQR extract L \n n = {} lda = {} \n error: {}", n,
            lda, e.what()));
    }

    // matrix_ops::print(A_inout, m, n, lda, "A_inout after get L");
}

template <typename T>
void panelQR(const common::CublasHandle& cublasHandle,
             const common::CusolverDnHandle& cusolverDnHandle, size_t m,
             size_t n, thrust::device_ptr<T> A_inout, size_t lda,
             thrust::device_ptr<T> R, size_t ldr, thrust::device_ptr<T> W,
             size_t ldw) {
    // tsqr, A_inout <- Q R
    matrix_ops::tsqr<T>(cublasHandle, m, n, A_inout, R, lda, ldr);

    try {
        size_t num_elements = m * n;
        thrust::for_each(thrust::device, thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(num_elements),
                         identity_minus_A_functor_2d<T>(A_inout, m, lda));
    } catch (std::exception& e) {
        throw std::runtime_error(
            fmt::format("Error in panelQR I-Q \n n = {} lda = {} \n error: {}",
                        n, lda, e.what()));
    }

    // matrix_ops::print(A_inout, m, n, lda, "A_inout after I-Q");

    // W = A_inout (a.k.a. I-Q)
    matrix_ops::matrix_copy<thrust::device_ptr<T>, thrust::device_ptr<T>, T>(
        A_inout, lda, W, ldw, m, n);

    // A <- "(I - Q) --> LU" [L]
    getIminusQL4panelQR(cusolverDnHandle, m, n, A_inout, lda);

    const auto alpha = static_cast<T>(1.0);
    if constexpr (std::is_same_v<T, double>) {
        hipblasDtrsm(cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                    HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha,
                    thrust::raw_pointer_cast(A_inout), lda,
                    thrust::raw_pointer_cast(W), ldw);
    } else if constexpr (std::is_same_v<T, float>) {
        hipblasStrsm(cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                    HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha,
                    thrust::raw_pointer_cast(A_inout), lda,
                    thrust::raw_pointer_cast(W), ldw);
    } else {
        throw std::runtime_error("Unsupported type.");
    }
}

template void getIminusQL4panelQR<float>(const common::CusolverDnHandle&,
                                         size_t, size_t,
                                         thrust::device_ptr<float>, size_t);
template void getIminusQL4panelQR<double>(const common::CusolverDnHandle&,
                                          size_t, size_t,
                                          thrust::device_ptr<double>, size_t);

template void panelQR<float>(const common::CublasHandle&,
                             const common::CusolverDnHandle&, size_t, size_t,
                             thrust::device_ptr<float>, size_t,
                             thrust::device_ptr<float>, size_t,
                             thrust::device_ptr<float>, size_t);
template void panelQR<double>(const common::CublasHandle&,
                              const common::CusolverDnHandle&, size_t, size_t,
                              thrust::device_ptr<double>, size_t,
                              thrust::device_ptr<double>, size_t,
                              thrust::device_ptr<double>, size_t);

}  // namespace sy2sb
}  // namespace internal
}  // namespace matrix_ops