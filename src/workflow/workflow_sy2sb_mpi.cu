#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mpi.h>

#include "gpu_handle_wrappers.h"
#include "log.h"
#include "matrix_ops.cuh"

template <typename T>
void run_workflow_sy2sb_mpi(size_t n, bool validate, int num_gpus = 1) {
    // Initialize MPI environment
    int provided;
    MPI_Init_thread(nullptr, nullptr, MPI_THREAD_MULTIPLE, &provided);
    if (provided < MPI_THREAD_MULTIPLE) {
        // Use regular logger before getting rank
        util::Logger::error("MPI does not support MPI_THREAD_MULTIPLE");
        MPI_Finalize();
        return;
    }

    // Get MPI rank and size
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Initialize MPI logger with rank information
    util::MpiLogger::init(util::Logger::is_verbose(), rank);

    if (rank == 0) {
        util::MpiLogger::print_environment_info();
        util::MpiLogger::println("Starting dist-evd-solver MPI version");
        util::MpiLogger::println("MPI initialized with {} processes", size);
    }

    // Initialize GPU devices for each MPI process
    int total_gpus;
    hipError_t cuda_err = hipGetDeviceCount(&total_gpus);
    if (cuda_err != hipSuccess) {
        util::MpiLogger::error("Failed to get CUDA device count: {}",
                              hipGetErrorString(cuda_err));
        MPI_Finalize();
        return;
    }

    // Bind each MPI process to specific GPU(s)
    // Strategy: round-robin assignment of GPUs to MPI processes
    int local_gpu_id = rank % total_gpus;
    cuda_err = hipSetDevice(local_gpu_id);
    if (cuda_err != hipSuccess) {
        util::MpiLogger::error("Failed to set CUDA device {}: {}", 
                              local_gpu_id, hipGetErrorString(cuda_err));
        MPI_Finalize();
        return;
    }

    // Verify GPU assignment
    int current_device;
    hipGetDevice(&current_device);
    util::MpiLogger::println("Bound to GPU {}", current_device);

    // Synchronize all processes after GPU initialization
    MPI_Barrier(MPI_COMM_WORLD);

    // TODO: Warm-up cuBLAS to avoid initialization overhead in timing
    {
        if (util::MpiLogger::is_verbose() && n <= 128) {
            util::MpiLogger::println("--- Performing cuBLAS warm-up ---");
        }
        // TODO: Implement cuBLAS warm-up
    }

    // Generate initial matrix A (symmetric) on rank 0
    auto A_h = thrust::host_vector<T>(n * n);
    if (rank == 0) {
        auto handle = common::CublasHandle();
        {
            auto A_d = matrix_ops::create_symmetric_random<T>(n, true);
            thrust::copy(A_d.begin(), A_d.end(), A_h.begin());
        }
        
        if (util::MpiLogger::is_verbose() && n <= 256) {
            matrix_ops::print(A_h.data(), n, n, n, "Initial matrix A");
        }
    }

    // TODO: Distribute matrix A across MPI processes

    // TODO: Perform symmetric-to-band (sy2sb) reduction using MPI

    // TODO: Gather results back to root process

    // TODO: Validate results if requested
    if (validate) {
        // TODO: Implement validation logic
    }

    // TODO: Cleanup and finalize MPI
    MPI_Finalize();
}

// Explicit template instantiation
template void run_workflow_sy2sb_mpi<float>(size_t n, bool validate,
                                            int num_gpus);
template void run_workflow_sy2sb_mpi<double>(size_t n, bool validate,
                                             int num_gpus);