#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hipsolver.h>
#include <fmt/format.h>

#include <cstddef>
#include <cstdlib>

#include "argh.h"
#include "log.h"
#include "workflow.cuh"

int main(int argc, char** argv) {
    // Parse command line arguments first
    argh::parser cmdl(argv);

    // Show help if requested
    if (cmdl[{"-h", "--help"}]) {
        fmt::print("Usage: {} [options]\n"
                  "Options:\n"
                  "  -n, --size <size>      Matrix size (default: 4)\n"
                  "  -m, --m <size>         Secondary matrix size (default: same as n)\n"
                  "  -g, --gpu-num <num>    Number of GPUs (default: 2)\n"
                  "  -nb, --nb <size>       Block size for recursion (default: 64)\n"
                  "  -b, --b <size>         Panel size (default: 16)\n"
                  "  --dist <mode>          Distribution: blockwise | cyclic (default: blockwise)\n"
                  "  -v, --verbose          Enable verbose output\n"
                  "  -t, --time             Enable timing output\n"
                  "  --validate             Enable result validation\n"
                  "  --double               Use double precision\n"
                  "  --float                Use single precision (default)\n"
                  "  -h, --help             Show this help message\n", argv[0]);
        return 0;
    }

    const bool verbose = cmdl[{"-v", "--verbose"}];
    const bool print_time = cmdl[{"-t", "--time"}];
    const bool validate = cmdl[{"--validate"}];

    auto n = (size_t)4;
    cmdl({"-n", "--size"}, 4) >> n;
    auto m = n;
    cmdl({"-m", "--m"}, n) >> m;

    auto gpu_num = 2;
    cmdl({"-g", "--gpu-num"}, 2) >> gpu_num;

    auto nb = (size_t)64;
    cmdl({"-nb", "--nb"}, 64) >> nb;

    auto b = (size_t)16;
    cmdl({"-b", "--b"}, 16) >> b;

    // Optional distribution selection (propagated via env var understood by sy2sb)
    std::string dist;
    cmdl({"--dist"}, std::string()) >> dist;
    if (!dist.empty()) {
        // Normalize aliases
        if (dist == "cyclic" || dist == "blockcyclic" || dist == "bc") {
            setenv("EVD_DIST", "cyclic", 1);
        } else if (dist == "blockwise" || dist == "bw") {
            setenv("EVD_DIST", "blockwise", 1);
        }
    }

    // Initialize logger (will be used by all ranks, but main info printed only by rank 0)
    util::Logger::init(verbose);
    util::Logger::init_timer(print_time);

    if (cmdl[{"--double"}]) {
        run_workflow_sy2sb_mpi<double>(n, validate, gpu_num, nb, b);
    } else if (cmdl[{"--float"}]) {
        run_workflow_sy2sb_mpi<float>(n, validate, gpu_num, nb, b);
    } else {
        run_workflow_sy2sb_mpi<float>(n, validate, gpu_num, nb, b);
    }

    return 0;
}
