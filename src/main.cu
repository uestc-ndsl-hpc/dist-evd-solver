#include "hip/hip_runtime.h"
#include <fmt/format.h>
#include <cstddef>

#include "argh.h"
#include "log.h"
#include "matrix_ops.cuh"

template <typename T>
void run_workflow(int n) {
    if constexpr (std::is_same_v<T, float>) {
        util::Logger::println("Using float precision");
    } else {
        util::Logger::println("Using double precision");
    }

    auto C = matrix_ops::create_symmetric_random<T>(n);

    if (util::Logger::is_verbose()) {
        matrix_ops::print(C, n, "Final Symmetric Matrix C");
    }
}

int main(int argc, char** argv) {
    argh::parser cmdl(argv);

    const bool verbose = cmdl[{"-v", "--verbose"}];
    util::Logger::init(verbose);
    const bool print_time = cmdl[{"-t", "--time"}];
    util::Logger::init_timer(print_time);
    util::Logger::println("Starting dist-evd-solver");

    auto n = (size_t) 4;
    cmdl({"-n", "--size"}, 4) >> n;

    if (cmdl[{"--double"}]) {
        util::Logger::println("Using double precision");
        run_workflow<double>(n);
    } else if (cmdl[{"--float"}]) {
        util::Logger::println("Using single precision");
        run_workflow<float>(n);
    } else {
        util::Logger::println("Using default precision");
        run_workflow<float>(n);
    }

    return 0;
}